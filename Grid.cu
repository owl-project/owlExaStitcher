#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2022-2022 Stefan Zellmann                                      //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include "Grid.h"
#include "Grid.cuh"

using namespace owl;

namespace exa {

  inline int __both__ iDivUp(int a, int b)
  {
    return (a + b - 1) / b;
  }

  __global__ void initGrid(float *cells, const vec3i dims)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= dims.x*size_t(dims.y)*dims.z)
      return;

    cells[threadID] = -1e30f;
  }

  // UMesh overload
  __global__ void buildGrid(float         *cells,
                            const vec4f   *vertices,
                            const int     *indices,
                            const size_t   numIndices,
                            const vec3i    dims,
                            const box3f    worldBounds)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= numIndices || indices[threadID] < 0)
      return;

    const vec4f V = vertices[indices[threadID]];

    const vec3i mc = projectOnGrid(vec3f(V),dims,worldBounds);

    cells[linearIndex(mc,dims)] = fmaxf(cells[linearIndex(mc,dims)],V.w);
  }

  // Gridlet overload
  __global__ void buildGrid(float         *cells,
                            const Gridlet *gridlets,
                            const size_t   numGridlets,
                            const vec3i    dims,
                            const box3f    worldBounds)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= numGridlets)
      return;

    const Gridlet &gridlet = gridlets[threadID];
    const vec3i numScalars = gridlet.dims+1;
    const vec3f halfCell = vec3f(1<<gridlet.level)*.5f;

    for (int z=0; z<numScalars.z; ++z) {
      for (int y=0; y<numScalars.y; ++y) {
        for (int x=0; x<numScalars.x; ++x) {
          const vec3f V = vec3f((gridlet.lower+vec3i(x,y,z)) * (1<<gridlet.level))
                        + halfCell;

          const vec3i mc = projectOnGrid(V,dims,worldBounds);

          const float value = gridlet.scalars[linearIndex(vec3i(x,y,z),numScalars)];

          cells[linearIndex(mc,dims)] = fmaxf(cells[linearIndex(mc,dims)],value);
        }
      }
    }
  }

  // AMR cell overload
  __global__ void buildGrid(float         *cells,
                            const AMRCell *amrCells,
                            const float   *amrScalars,
                            const size_t   numAmrCells,
                            const vec3i    dims,
                            const box3f    worldBounds)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= numAmrCells)
      return;

    const AMRCell &cell = amrCells[threadID];

    const vec3f V = vec3f(cell.pos+vec3i(1<<cell.level) - (vec3i(1<<cell.level)/2));

    const vec3i mc = projectOnGrid(V,dims,worldBounds);

    const float value = amrScalars[threadID];

    cells[linearIndex(mc,dims)] = fmaxf(cells[linearIndex(mc,dims)],value);
  }

  void Grid::build(OWLContext  owl,
                   OWLBuffer   vertices,
                   OWLBuffer   indices,
                   OWLBuffer   gridlets,
                   OWLBuffer   amrCells,
                   OWLBuffer   amrScalars,
                   const vec3i numMCs,
                   const box3f bounds)
  {
    dims        = numMCs;
    worldBounds = bounds;

    cells = owlDeviceBufferCreate(owl, OWL_FLOAT,
                                  dims.x*size_t(dims.y)*dims.z,
                                  nullptr);

    // Init with small floats
    {
      size_t numThreads = 1024;
      size_t numMCs = dims.x*size_t(dims.y)*dims.z;
      initGrid<<<iDivUp(numMCs, numThreads), numThreads>>>
        ((float *)owlBufferGetPointer(cells,0),dims);
    }

    // Add contrib from uelems
    if (vertices && indices) {
      size_t numThreads = 1024;
      size_t numIndices = owlBufferSizeInBytes(indices)/sizeof(int);
      std::cout << "DDA grid: adding " << numIndices << " uelems\n";
      buildGrid<<<iDivUp(numIndices, numThreads), numThreads>>>(
        (float *)owlBufferGetPointer(cells,0),
        (const vec4f *)owlBufferGetPointer(vertices,0),
        (const int *)owlBufferGetPointer(indices,0),
        numIndices,dims,worldBounds);
    }

    // Add contrib from gridlets
    if (gridlets) {
      size_t numThreads = 1024;
      size_t numGridlets = owlBufferSizeInBytes(gridlets)/sizeof(Gridlet);
      std::cout << "DDA grid: adding " << numGridlets << " gridlets\n";
      buildGrid<<<iDivUp(numGridlets, numThreads), numThreads>>>(
        (float *)owlBufferGetPointer(cells,0),
        (const Gridlet *)owlBufferGetPointer(gridlets,0),
        numGridlets,dims,worldBounds);
    }

    // Add contrib from AMR cells
    if (amrCells && amrScalars) {
      size_t numThreads = 1024;
      size_t numAmrCells = owlBufferSizeInBytes(amrCells)/sizeof(AMRCell);
      std::cout << "DDA grid: adding " << numAmrCells << " AMR cells (non-dual!)\n";
      buildGrid<<<iDivUp(numAmrCells, numThreads), numThreads>>>(
        (float *)owlBufferGetPointer(cells,0),
        (const AMRCell *)owlBufferGetPointer(amrCells,0),
        (const float *)owlBufferGetPointer(amrScalars,0),
        numAmrCells,dims,worldBounds);
    }
  }
} // ::exa

// vim: sw=2:expandtab:softtabstop=2:ts=2:cino=\:0g0t0

