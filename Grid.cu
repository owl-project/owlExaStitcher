#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2022-2022 Stefan Zellmann                                      //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "sampler/AMRCellSampler.h"
#include "sampler/ExaBrickSampler.h"
#include "sampler/ExaStitchSampler.h"
#include "sampler/QuickClustersSampler.h"
#include "common.h"
#include "LaunchParams.h" // for MacroCellGeom (dependency should be removed eventually)
#include "Grid.h"
#include "Grid.cuh"
#include "atomicOp.cuh"

using namespace owl;

namespace exa {

  template<typename T>
  inline T __both__ iDivUp(T a, T b)
  {
    return (a + b - 1) / b;
  }

  template<typename T>
  inline T __both__ iRoundUp(T a, T b)
  {
    return iDivUp(a,b) * b;
  }

  __device__ inline void updateMC(const vec3i    mcID,
                                  const vec3i    gridDims,
                                  const range1f  valueRange,
                                  range1f       *valueRanges)
  {
    atomicMin(&valueRanges[linearIndex(mcID,gridDims)].lower,valueRange.lower);
    atomicMax(&valueRanges[linearIndex(mcID,gridDims)].upper,valueRange.upper);
  }

  __device__ inline void updateMC(const vec3i  mcID,
                                  const vec3i  gridDims,
                                  const float  value,
                                  range1f     *valueRanges)
  {
    updateMC(mcID,gridDims,range1f{value,value},valueRanges);
  }


  __global__ void initGrid(range1f *valueRanges, const vec3i dims)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= dims.x*size_t(dims.y)*dims.z)
      return;

    valueRanges[threadID].lower = +1e30f;
    valueRanges[threadID].upper = -1e30f;
  }

  // UMesh overload
  template <int NumVertsMax=8>
  __global__ void buildGrid(range1f       *valueRanges,
                            const vec4f   *vertices,
                            const int     *indices,
                            const size_t   numElems,
                            const vec3i    dims,
                            const box3f    worldBounds)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= numElems)
      return;

    const int *I = &indices[threadID*NumVertsMax];

    box3f cellBounds;
    range1f valueRange{+1e30f,-1e30f};

    for (int i=0; i<NumVertsMax; ++i) {
      if (I[i] < 0)
        break;

      const vec4f V = vertices[I[i]];
      if (!isnan(V.w)) {
        cellBounds.extend(vec3f(V));
        valueRange.lower = fminf(valueRange.lower,V.w);
        valueRange.upper = fmaxf(valueRange.upper,V.w);
      }
    }

    const vec3i loMC = projectOnGrid(cellBounds.lower,dims,worldBounds);
    const vec3i upMC = projectOnGrid(cellBounds.upper,dims,worldBounds);
    //printf("%i,%i,%i -- %i,%i,%i,\n",
    //       loMC.x,loMC.y,loMC.z,
    //       upMC.x,upMC.y,upMC.z);

    for (int mcz=loMC.z; mcz<=upMC.z; ++mcz) {
      for (int mcy=loMC.y; mcy<=upMC.y; ++mcy) {
        for (int mcx=loMC.x; mcx<=upMC.x; ++mcx) {
          const vec3i mcID(mcx,mcy,mcz);
          updateMC(mcID,dims,valueRange,valueRanges);
        }
      }
    }
  }

  // Gridlet overload
  __global__ void getMaxGridletSize(const Gridlet *gridlets,
                                    const size_t   numGridlets,
                                    vec3i         *maxGridletSize)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= numGridlets)
      return;

    ::atomicMax(&maxGridletSize->x,gridlets[threadID].dims.x);
    ::atomicMax(&maxGridletSize->y,gridlets[threadID].dims.y);
    ::atomicMax(&maxGridletSize->z,gridlets[threadID].dims.z);
  }

  __global__ void buildGrid(range1f       *valueRanges,
                            const Gridlet *gridlets,
                            const float   *scalars,
                            const size_t   numGridlets,
                            const vec3i    maxGridletSize,
                            const vec3i    dims,
                            const box3f    worldBounds)
  {
    size_t gridletID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (gridletID >= numGridlets)
      return;

    const Gridlet &gridlet = gridlets[gridletID];

    int xyz = blockIdx.y * blockDim.y + threadIdx.y;
    int x = xyz%maxGridletSize.x;
    int y = xyz/maxGridletSize.x%maxGridletSize.y;
    int z = xyz/(maxGridletSize.x*maxGridletSize.y);
    int dimX = min(maxGridletSize.x,gridlet.dims.x);
    int dimY = min(maxGridletSize.y,gridlet.dims.y);
    int dimZ = min(maxGridletSize.z,gridlet.dims.z);

    if (x >= dimX || y >= dimY || z >= dimZ)
      return;

    const box3f &gridletBounds = gridlet.getBounds();

    const vec3f mcSize(worldBounds.size() / vec3f(dims));
    const vec3i loMC = projectOnGrid(gridletBounds.lower,dims,worldBounds);
    const vec3i upMC = projectOnGrid(gridletBounds.upper,dims,worldBounds);

    const vec3i numScalars = gridlet.dims+1;

    for (int mcz=loMC.z; mcz<=upMC.z; ++mcz) {
      for (int mcy=loMC.y; mcy<=upMC.y; ++mcy) {
        for (int mcx=loMC.x; mcx<=upMC.x; ++mcx) {
          const vec3i mcID(mcx,mcy,mcz);
          const box3f mcBounds(worldBounds.lower+vec3f(mcID)*mcSize,
                               worldBounds.lower+vec3f(mcID+1)*mcSize);


          range1f valueRange{+1e30f,-1e30f};
          /*for (int z=0; z<gridlet.dims.z; ++z)*/ {
            /*for (int y=0; y<gridlet.dims.y; ++y)*/ {
              /*for (int x=0; x<gridlet.dims.x; ++x)*/ {
                const float cellWidth = 1<<gridlet.level;
                const box3f cellBounds(vec3f((gridlet.lower+vec3i(x,y,z)) * cellWidth) + .5f*cellWidth,
                                       vec3f((gridlet.lower+vec3i(x+1,y+1,z+1)) * cellWidth) + .5f*cellWidth);
                if (mcBounds.overlaps(cellBounds)) {
                  vec3i imin(x,y,z);
                  vec3i imax(x+1,y+1,z+1);


                  float f1 = scalars[gridlet.begin+linearIndex(vec3i(imin.x,imin.y,imin.z),numScalars)];
                  float f2 = scalars[gridlet.begin+linearIndex(vec3i(imax.x,imin.y,imin.z),numScalars)];
                  float f3 = scalars[gridlet.begin+linearIndex(vec3i(imin.x,imax.y,imin.z),numScalars)];
                  float f4 = scalars[gridlet.begin+linearIndex(vec3i(imax.x,imax.y,imin.z),numScalars)];

                  float f5 = scalars[gridlet.begin+linearIndex(vec3i(imin.x,imin.y,imax.z),numScalars)];
                  float f6 = scalars[gridlet.begin+linearIndex(vec3i(imax.x,imin.y,imax.z),numScalars)];
                  float f7 = scalars[gridlet.begin+linearIndex(vec3i(imin.x,imax.y,imax.z),numScalars)];
                  float f8 = scalars[gridlet.begin+linearIndex(vec3i(imax.x,imax.y,imax.z),numScalars)];

                  if (!isnan(f1)) {
                    valueRange.lower = fminf(valueRange.lower,f1);
                    valueRange.upper = fmaxf(valueRange.upper,f1);
                  }

                  if (!isnan(f2)) {
                    valueRange.lower = fminf(valueRange.lower,f2);
                    valueRange.upper = fmaxf(valueRange.upper,f2);
                  }

                  if (!isnan(f3))  {
                    valueRange.lower = fminf(valueRange.lower,f3);
                    valueRange.upper = fmaxf(valueRange.upper,f3);
                  }

                  if (!isnan(f4)) {
                    valueRange.lower = fminf(valueRange.lower,f4);
                    valueRange.upper = fmaxf(valueRange.upper,f4);
                  }

                  if (!isnan(f5)) {
                    valueRange.lower = fminf(valueRange.lower,f5);
                    valueRange.upper = fmaxf(valueRange.upper,f5);
                  }

                  if (!isnan(f6)) {
                    valueRange.lower = fminf(valueRange.lower,f6);
                    valueRange.upper = fmaxf(valueRange.upper,f6);
                  }

                  if (!isnan(f7)) {
                    valueRange.lower = fminf(valueRange.lower,f7);
                    valueRange.upper = fmaxf(valueRange.upper,f7);
                  }

                  if (!isnan(f8)) {
                    valueRange.lower = fminf(valueRange.lower,f8);
                    valueRange.upper = fmaxf(valueRange.upper,f8);
                  }

                }
              }
            }
          }
          updateMC(mcID,dims,valueRange,valueRanges);
        }
      }
    }
  }

  // AMR cell overload
  __global__ void buildGrid(range1f       *valueRanges,
                            const AMRCell *cells,
                            const float   *scalars,
                            const size_t   numAmrCells,
                            const vec3i    dims,
                            const box3f    worldBounds)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= numAmrCells)
      return;

    const AMRCell &cell = cells[threadID];

    vec3i lower = cell.pos;
    vec3i upper = lower + (1<<cell.level);

    const vec3f halfCell = vec3f(1<<cell.level)*.5f;

    const vec3i loMC = projectOnGrid(vec3f(lower)-halfCell,dims,worldBounds);
    const vec3i upMC = projectOnGrid(vec3f(upper)+halfCell,dims,worldBounds);

    const float value = scalars[threadID];

    if (!isnan(value)) {
      for (int mcz=loMC.z; mcz<=upMC.z; ++mcz) {
        for (int mcy=loMC.y; mcy<=upMC.y; ++mcy) {
          for (int mcx=loMC.x; mcx<=upMC.x; ++mcx) {
            const vec3i mcID(mcx,mcy,mcz);
            updateMC(mcID,dims,value,valueRanges);
          }
        }
      }
    }
  }

  // ExaBrick overload
  __global__ void buildGrid(range1f      *valueRanges,
                            const ABR    *abrs,
                            const size_t  numABRs,
                            const vec3i   dims,
                            const box3f   worldBounds)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= numABRs)
      return;

    const box3f domain = abrs[threadID].domain;
    const range1f valueRange = abrs[threadID].valueRange;

    const vec3f mcSize(worldBounds.size() / vec3f(dims));
    const vec3i loMC = projectOnGrid(domain.lower,dims,worldBounds);
    const vec3i upMC = projectOnGrid(domain.upper,dims,worldBounds);

    for (int mcz=loMC.z; mcz<=upMC.z; ++mcz) {
      for (int mcy=loMC.y; mcy<=upMC.y; ++mcy) {
        for (int mcx=loMC.x; mcx<=upMC.x; ++mcx) {
          const vec3i mcID(mcx,mcy,mcz);
          updateMC(mcID,dims,valueRange,valueRanges);
        }
      }
    }
  }

  void Grid::build(OWLContext         owl,
                   AMRCellSampler::SP sampler,
                   const owl::vec3i   numMCs,
                   const owl::box3f   bounds)
  {
    dims        = numMCs;
    worldBounds = bounds;

    valueRanges = owlDeviceBufferCreate(owl, OWL_USER_TYPE(range1f),
                                        dims.x*size_t(dims.y)*dims.z,
                                        nullptr);

    // Init with small floats
    {
      size_t numThreads = 1024;
      size_t numMCs = dims.x*size_t(dims.y)*dims.z;
      initGrid<<<(uint32_t)iDivUp(numMCs, numThreads), (uint32_t)numThreads>>>
        ((range1f *)owlBufferGetPointer(valueRanges,0),dims);

      // pre-allocating max-opacity buffer
      maxOpacities = owlDeviceBufferCreate(owl, OWL_FLOAT, numMCs, nullptr);
    }

    // Add contrib from AMR cells
    {
      size_t numThreads = 1024;
      size_t numAmrCells = owlBufferSizeInBytes(sampler->cellBuffer)/sizeof(AMRCell);
      std::cout << "DDA grid: adding " << numAmrCells << " AMR cells (non-dual!)\n";
      buildGrid<<<(uint32_t)iDivUp(numAmrCells, numThreads), (uint32_t)numThreads>>>(
        (range1f *)owlBufferGetPointer(valueRanges,0),
        (const AMRCell *)owlBufferGetPointer(sampler->cellBuffer,0),
        (const float *)owlBufferGetPointer(sampler->scalarBuffer,0),
        numAmrCells,dims,worldBounds);
      hipDeviceSynchronize();
      std::cout << hipGetErrorString(hipGetLastError()) << '\n';
    }

    // init device traversable for DDA
#ifdef EXA_STITCH_MIRROR_EXAJET
    deviceTraversable.traversable.dims = dims;
    deviceTraversable.traversable.bounds = worldBounds;
#else
    deviceTraversable.dims = dims;
    deviceTraversable.bounds = worldBounds;
#endif

  }

  void Grid::build(OWLContext          owl,
                   ExaBrickSampler::SP sampler,
                   const owl::vec3i    numMCs,
                   const owl::box3f    bounds)
  {
    dims        = numMCs;
    worldBounds = bounds;

    valueRanges = owlDeviceBufferCreate(owl, OWL_USER_TYPE(range1f),
                                        dims.x*size_t(dims.y)*dims.z,
                                        nullptr);

    // Init with small floats
    {
      size_t numThreads = 1024;
      size_t numMCs = dims.x*size_t(dims.y)*dims.z;
      initGrid<<<(uint32_t)iDivUp(numMCs, numThreads), (uint32_t)numThreads>>>
        ((range1f *)owlBufferGetPointer(valueRanges,0),dims);

      // pre-allocating max-opacity buffer
      maxOpacities = owlDeviceBufferCreate(owl, OWL_FLOAT, numMCs, nullptr);
    }

    // Add contrib from ExaBricks
    if (0) { // ABR grid projection
      double tfirst = getCurrentTime();
      size_t numThreads = 1024;
      size_t numABRs = owlBufferSizeInBytes(sampler->abrBuffer)/sizeof(ABR);
      std::cout << "DDA grid: adding " << numABRs << " ExaBrick ABRs\n";
      buildGrid<<<(uint32_t)iDivUp(numABRs, numThreads), (uint32_t)numThreads>>>(
        (range1f *)owlBufferGetPointer(valueRanges,0),
        (const ABR *)owlBufferGetPointer(sampler->abrBuffer,0),
        numABRs,dims,worldBounds);

      std::cout << hipGetErrorString(hipGetLastError()) << '\n';
      double tlast = getCurrentTime();
      std::cout << tlast-tfirst << '\n';
    } else { // project cells; for now on the CPU
      std::vector<range1f> hValueRanges(dims.x*size_t(dims.y)*dims.z);
      std::fill(hValueRanges.begin(),
                hValueRanges.end(),
                range1f{1e30f,-1e30f});

      for (size_t i=0; i<sampler->model->bricks.size(); ++i) {
        const ExaBrick &brick = sampler->model->bricks[i];
        for (int z=0; z<brick.size.z; ++z) {
          for (int y=0; y<brick.size.y; ++y) {
            for (int x=0; x<brick.size.x; ++x) {
              vec3i index3(x,y,z);
              int idx = brick.getIndexIndex(index3);
              const float value = sampler->model->scalars[idx];

              vec3i lower = brick.lower + index3*(1<<brick.level);
              vec3i upper = lower + (1<<brick.level);

              const vec3f halfCell = vec3f((float)(1<<brick.level))*.5f;

              const vec3i loMC = projectOnGrid(vec3f(lower)-halfCell,dims,worldBounds);
              const vec3i upMC = projectOnGrid(vec3f(upper)+halfCell,dims,worldBounds);

              for (int mcz=loMC.z; mcz<=upMC.z; ++mcz) {
                for (int mcy=loMC.y; mcy<=upMC.y; ++mcy) {
                  for (int mcx=loMC.x; mcx<=upMC.x; ++mcx) {
                    const vec3i mcID(mcx,mcy,mcz);
                    hValueRanges[linearIndex(mcID,dims)].lower
                      = std::min(hValueRanges[linearIndex(mcID,dims)].lower,value);
                    hValueRanges[linearIndex(mcID,dims)].upper
                      = std::max(hValueRanges[linearIndex(mcID,dims)].upper,value);
                  }
                }
              }
            }
          }
        }

        std::cout << '(' << (i+1) << '/' << sampler->model->bricks.size() << ")\r";
      }

      owlBufferRelease(valueRanges);
      valueRanges = owlDeviceBufferCreate(owl, OWL_USER_TYPE(range1f),
                                          hValueRanges.size(),
                                          hValueRanges.data());
    }

    // init device traversable for DDA
#ifdef EXA_STITCH_MIRROR_EXAJET
    deviceTraversable.traversable.dims = dims;
    deviceTraversable.traversable.bounds = worldBounds;
#else
    deviceTraversable.dims = dims;
    deviceTraversable.bounds = worldBounds;
#endif

  }

  void Grid::build(OWLContext           owl,
                   ExaStitchSampler::SP sampler,
                   const owl::vec3i     numMCs,
                   const owl::box3f     bounds)
  {
    dims        = numMCs;
    worldBounds = bounds;

    valueRanges = owlDeviceBufferCreate(owl, OWL_USER_TYPE(range1f),
                                        dims.x*size_t(dims.y)*dims.z,
                                        nullptr);

    // Init with small floats
    {
      size_t numThreads = 1024;
      size_t numMCs = dims.x*size_t(dims.y)*dims.z;
      initGrid<<<(uint32_t)iDivUp(numMCs, numThreads), (uint32_t)numThreads>>>
        ((range1f *)owlBufferGetPointer(valueRanges,0),dims);

      // pre-allocating max-opacity buffer
      maxOpacities = owlDeviceBufferCreate(owl, OWL_FLOAT, numMCs, nullptr);
    }

    // Add contrib from uelems
#ifdef EXA_STITCH_SEPARATE_INDEX_BUFFERS_PER_UELEM
    for (int i=0; i<4; ++i) {
      if (sampler->vertexBuffer && sampler->indexBuffers[i])
      {
        if (owlBufferSizeInBytes(sampler->indexBuffers[i])==0)
          continue;

        size_t numThreads = 1024;
        // TODO: use some template magic here (?)
        if (i==0) {
          size_t numElems = owlBufferSizeInBytes(sampler->indexBuffers[i])/sizeof(int[4]);
          std::cout << "DDA grid: adding " << numElems << " uelems of type " << i << '\n';
          buildGrid<4><<<iDivUp(numElems, numThreads), numThreads>>>(
            (range1f *)owlBufferGetPointer(valueRanges,0),
            (const vec4f *)owlBufferGetPointer(sampler->vertexBuffer,0),
            (const int *)owlBufferGetPointer(sampler->indexBuffers[i],0),
            numElems,dims,worldBounds);
        }
        else if (i==1) {
          size_t numElems = owlBufferSizeInBytes(sampler->indexBuffers[i])/sizeof(int[5]);
          std::cout << "DDA grid: adding " << numElems << " uelems of type " << i << '\n';
          buildGrid<5><<<iDivUp(numElems, numThreads), numThreads>>>(
            (range1f *)owlBufferGetPointer(valueRanges,0),
            (const vec4f *)owlBufferGetPointer(sampler->vertexBuffer,0),
            (const int *)owlBufferGetPointer(sampler->indexBuffers[i],0),
            numElems,dims,worldBounds);
        }
        else if (i==2) {
          size_t numElems = owlBufferSizeInBytes(sampler->indexBuffers[i])/sizeof(int[6]);
          std::cout << "DDA grid: adding " << numElems << " uelems of type " << i << '\n';
          buildGrid<6><<<iDivUp(numElems, numThreads), numThreads>>>(
            (range1f *)owlBufferGetPointer(valueRanges,0),
            (const vec4f *)owlBufferGetPointer(sampler->vertexBuffer,0),
            (const int *)owlBufferGetPointer(sampler->indexBuffers[i],0),
            numElems,dims,worldBounds);
        }
        else if (i==3) {
          size_t numElems = owlBufferSizeInBytes(sampler->indexBuffers[i])/sizeof(int[8]);
          std::cout << "DDA grid: adding " << numElems << " uelems of type " << i << '\n';
          buildGrid<8><<<iDivUp(numElems, numThreads), numThreads>>>(
            (range1f *)owlBufferGetPointer(valueRanges,0),
            (const vec4f *)owlBufferGetPointer(sampler->vertexBuffer,0),
            (const int *)owlBufferGetPointer(sampler->indexBuffers[i],0),
            numElems,dims,worldBounds);
        }
        hipDeviceSynchronize();
        std::cout << hipGetErrorString(hipGetLastError()) << '\n';
      }
    }
#else
    if (sampler->vertexBuffer && sampler->indexBuffer)
    {
      size_t numThreads = 1024;
      size_t numElems = owlBufferSizeInBytes(sampler->indexBuffer)/sizeof(int[8]);
      std::cout << "DDA grid: adding " << numElems << " uelems\n";
      buildGrid<8><<<iDivUp(numElems, numThreads), numThreads>>>(
        (range1f *)owlBufferGetPointer(valueRanges,0),
        (const vec4f *)owlBufferGetPointer(sampler->vertexBuffer,0),
        (const int *)owlBufferGetPointer(sampler->indexBuffer,0),
        numElems,dims,worldBounds);
      hipDeviceSynchronize();
      std::cout << hipGetErrorString(hipGetLastError()) << '\n';
    }
#endif

    // Add contrib from gridlets
    if (sampler->gridletBuffer && sampler->gridletScalarBuffer)
    {
      double tfirst = getCurrentTime();
      size_t numThreads = 1024;
      size_t numGridlets = owlBufferSizeInBytes(sampler->gridletBuffer)/sizeof(Gridlet);
      std::cout << "DDA grid: adding " << numGridlets << " gridlets\n";
      vec3i *maxGridletSize;
      hipMalloc(&maxGridletSize,sizeof(vec3i));
      vec3i init = 0;
      hipMemcpy(maxGridletSize,&init,sizeof(init),hipMemcpyHostToDevice);

      getMaxGridletSize<<<(uint32_t)iDivUp(numGridlets, numThreads), (uint32_t)numThreads>>>(
        (const Gridlet *)owlBufferGetPointer(sampler->gridletBuffer,0),
        numGridlets,maxGridletSize);

      vec3i hMaxGridletSize;
      hipMemcpy(&hMaxGridletSize,maxGridletSize,sizeof(hMaxGridletSize),
                 hipMemcpyDeviceToHost);

      dim3 gridDims((int)numGridlets,
                    (int)(hMaxGridletSize.x*hMaxGridletSize.y*hMaxGridletSize.z));
      dim3 blockDims(64,16);
      dim3 numBlocks(iDivUp(gridDims.x,blockDims.x),
                     iDivUp(gridDims.y,blockDims.y));

      buildGrid<<<numBlocks, blockDims>>>(
        (range1f *)owlBufferGetPointer(valueRanges,0),
        (const Gridlet *)owlBufferGetPointer(sampler->gridletBuffer,0),
        (const float *)owlBufferGetPointer(sampler->gridletScalarBuffer,0),
        numGridlets,hMaxGridletSize,dims,worldBounds);

      hipFree(maxGridletSize);
      hipDeviceSynchronize();
      std::cout << hipGetErrorString(hipGetLastError()) << '\n';
      double tlast = getCurrentTime();
      std::cout << tlast-tfirst << '\n';
    }

    // init device traversable for DDA
#ifdef EXA_STITCH_MIRROR_EXAJET
    deviceTraversable.traversable.dims = dims;
    deviceTraversable.traversable.bounds = worldBounds;
#else
    deviceTraversable.dims = dims;
    deviceTraversable.bounds = worldBounds;
#endif

  }

  void Grid::build(OWLContext           owl,
                   QuickClustersSampler::SP sampler,
                   const owl::vec3i     numMCs,
                   const owl::box3f     bounds)
  {
    dims        = numMCs;
    worldBounds = bounds;

    valueRanges = owlDeviceBufferCreate(owl, OWL_USER_TYPE(range1f),
                                        dims.x*size_t(dims.y)*dims.z,
                                        nullptr);

    // Init with small floats
    {
      size_t numThreads = 1024;
      size_t numMCs = dims.x*size_t(dims.y)*dims.z;
      initGrid<<<(uint32_t)iDivUp(numMCs, numThreads), (uint32_t)numThreads>>>
        ((range1f *)owlBufferGetPointer(valueRanges,0),dims);

      // pre-allocating max-opacity buffer
      maxOpacities = owlDeviceBufferCreate(owl, OWL_FLOAT, numMCs, nullptr);
    }

    // Add contrib from uelems
    {
      size_t numThreads = 1024;
      size_t numElems = owlBufferSizeInBytes(sampler->indexBuffer)/sizeof(int[8]);
      std::cout << "DDA grid: adding " << numElems << " uelems\n";
      buildGrid<<<(uint32_t)iDivUp(numElems, numThreads), (uint32_t)numThreads>>>(
        (range1f *)owlBufferGetPointer(valueRanges,0),
        (const vec4f *)owlBufferGetPointer(sampler->vertexBuffer,0),
        (const int *)owlBufferGetPointer(sampler->indexBuffer,0),
        numElems,dims,worldBounds);
      hipDeviceSynchronize();
      std::cout << hipGetErrorString(hipGetLastError()) << '\n';
    }

    // init device traversable for DDA
#ifdef EXA_STITCH_MIRROR_EXAJET
    deviceTraversable.traversable.dims = dims;
    deviceTraversable.traversable.bounds = worldBounds;
#else
    deviceTraversable.dims = dims;
    deviceTraversable.bounds = worldBounds;
#endif

  }

  bool Grid::buildOptixBVH(OWLContext owl, OWLModule module)
  {
    // build BVH (tarversal method a)
    OWLVarDecl geomVars[]
    = {
       { "dims", OWL_INT3, OWL_OFFSETOF(MacroCellGeom,dims) },
       { "spacing", OWL_FLOAT3, OWL_OFFSETOF(MacroCellGeom,spacing) },
       { "origin", OWL_FLOAT3, OWL_OFFSETOF(MacroCellGeom,origin) },
       { "maxOpacities", OWL_BUFPTR, OWL_OFFSETOF(MacroCellGeom,maxOpacities) },
       { nullptr /* sentinel to mark end of list */ }
    };

    const vec3f spacing(worldBounds.size() / vec3f(dims));

    geomType = owlGeomTypeCreate(owl, OWL_GEOM_USER, sizeof(MacroCellGeom), geomVars, -1);
    owlGeomTypeSetBoundsProg   (geomType, module, "MacroCellGeomBounds");
    owlGeomTypeSetIntersectProg(geomType, RADIANCE_RAY_TYPE, module, "MacroCellGeomIsect");
    owlGeomTypeSetClosestHit   (geomType, RADIANCE_RAY_TYPE, module, "MacroCellGeomCH");
    OWLGeom geom = owlGeomCreate(owl, geomType);
    owlGeomSetPrimCount(geom, size_t(dims.x)*size_t(dims.y)*size_t(dims.z));
    owlGeomSet3i(geom,"dims", dims.x, dims.y, dims.z);
    owlGeomSet3f(geom,"spacing", spacing.x, spacing.y, spacing.z);
    owlGeomSet3f(geom,"origin", worldBounds.lower.x, worldBounds.lower.y, worldBounds.lower.z);
    owlGeomSetBuffer(geom,"maxOpacities", maxOpacities);

    owlBuildPrograms(owl);

    blas = owlUserGeomGroupCreate(owl, 1, &geom);
#ifdef EXA_STITCH_MIRROR_EXAJET
    owlGroupBuildAccel(blas);
    tlas = owlInstanceGroupCreate(owl, 2);
    owlInstanceGroupSetChild(tlas, 0, blas);
    owlInstanceGroupSetChild(tlas, 1, blas);
#else
    owlGroupBuildAccel(blas);
    tlas = owlInstanceGroupCreate(owl, 1);
    owlInstanceGroupSetChild(tlas, 0, blas);
    owlGroupBuildAccel(tlas);
#endif

    return true;
  }

  __global__ void computeMaxOpacitiesGPU(float         *maxOpacities,
                                         const range1f *valueRanges,
                                         const vec4f   *colorMap,
                                         size_t         numMCs,
                                         size_t         numColors,
                                         range1f        xfRange)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= numMCs)
      return;

    range1f valueRange = valueRanges[threadID];

    if (valueRange.upper < valueRange.lower) {
      maxOpacities[threadID] = 0.f;
      return;
    }

    valueRange.lower -= xfRange.lower;
    valueRange.lower /= xfRange.upper-xfRange.lower;
    valueRange.upper -= xfRange.lower;
    valueRange.upper /= xfRange.upper-xfRange.lower;

    int lo = clamp(int(valueRange.lower*(numColors-1)),0,(int)numColors-1);
    int hi = clamp(int(valueRange.upper*(numColors-1))+1,0,(int)numColors-1);

    float maxOpacity = 0.f;
    for (int i=lo; i<=hi; ++i) {
      maxOpacity = fmaxf(maxOpacity,colorMap[i].w);
    }
    maxOpacities[threadID] = maxOpacity;
  }

  void Grid::computeMaxOpacities(OWLContext owl, OWLBuffer colorMap, range1f xfRange)
  {
    size_t numMCs = dims.x*size_t(dims.y)*dims.z;
    size_t numColors = owlBufferSizeInBytes(colorMap)/sizeof(vec4f);

    size_t numThreads = 1024;
    computeMaxOpacitiesGPU<<<(uint32_t)iDivUp(numMCs, numThreads), (uint32_t)numThreads>>>(
      (float *)owlBufferGetPointer(maxOpacities,0),
      (const range1f *)owlBufferGetPointer(valueRanges,0),
      (const vec4f *)owlBufferGetPointer(colorMap,0),
      numMCs,numColors,xfRange);

#if EXA_STITCH_EXA_BRICK_TRAVERSAL_MODE == MC_BVH_TRAVERSAL
    owlGroupBuildAccel(blas);
    owlGroupBuildAccel(tlas);
#endif
  }

} // ::exa

// vim: sw=2:expandtab:softtabstop=2:ts=2:cino=\:0g0t0

