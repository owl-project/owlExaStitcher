#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2022-2022 Stefan Zellmann                                      //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include "Grid.h"
#include "Grid.cuh"
#include "atomicOp.cuh"
#include "AMRCellModel.h"
#include "ExaBrickModel.h"
#include "ExaStitchModel.h"

using namespace owl;

namespace exa {


  inline int __both__ iDivUp(int a, int b)
  {
    return (a + b - 1) / b;
  }

  inline int __both__ iRoundUp(int a, int b)
  {
    return iDivUp(a,b) * b;
  }

  __device__ inline void updateMC(const vec3i    mcID,
                                  const vec3i    gridDims,
                                  const range1f  valueRange,
                                  range1f       *valueRanges)
  {
    atomicMin(&valueRanges[linearIndex(mcID,gridDims)].lower,valueRange.lower);
    atomicMax(&valueRanges[linearIndex(mcID,gridDims)].upper,valueRange.upper);
  }

  __device__ inline void updateMC(const vec3i  mcID,
                                  const vec3i  gridDims,
                                  const float  value,
                                  range1f     *valueRanges)
  {
    updateMC(mcID,gridDims,range1f{value,value},valueRanges);
  }


  __global__ void initGrid(range1f *valueRanges, const vec3i dims)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= dims.x*size_t(dims.y)*dims.z)
      return;

    valueRanges[threadID].lower = +1e30f;
    valueRanges[threadID].upper = -1e30f;
  }

  // UMesh overload
  __global__ void buildGrid(range1f       *valueRanges,
                            const vec4f   *vertices,
                            const int     *indices,
                            const size_t   numElems,
                            const vec3i    dims,
                            const box3f    worldBounds)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= numElems)
      return;

    const int *I = &indices[threadID*8];

    box3f cellBounds;
    range1f valueRange{+1e30f,-1e30f};

    for (int i=0; i<8; ++i) {
      if (I[i] < 0)
        break;

      const vec4f V = vertices[I[i]];
      if (!isnan(V.w)) {
        cellBounds.extend(vec3f(V));
        valueRange.lower = fminf(valueRange.lower,V.w);
        valueRange.upper = fmaxf(valueRange.upper,V.w);
      }
    }

    const vec3i loMC = projectOnGrid(cellBounds.lower,dims,worldBounds);
    const vec3i upMC = projectOnGrid(cellBounds.upper,dims,worldBounds);
    //printf("%i,%i,%i -- %i,%i,%i,\n",
    //       loMC.x,loMC.y,loMC.z,
    //       upMC.x,upMC.y,upMC.z);

    for (int mcz=loMC.z; mcz<=upMC.z; ++mcz) {
      for (int mcy=loMC.y; mcy<=upMC.y; ++mcy) {
        for (int mcx=loMC.x; mcx<=upMC.x; ++mcx) {
          const vec3i mcID(mcx,mcy,mcz);
          updateMC(mcID,dims,valueRange,valueRanges);
        }
      }
    }
  }

  // Gridlet overload
  __global__ void getMaxGridletSize(const Gridlet *gridlets,
                                    const size_t   numGridlets,
                                    vec3i         *maxGridletSize)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= numGridlets)
      return;

    ::atomicMax(&maxGridletSize->x,gridlets[threadID].dims.x);
    ::atomicMax(&maxGridletSize->y,gridlets[threadID].dims.y);
    ::atomicMax(&maxGridletSize->z,gridlets[threadID].dims.z);
  }

  __global__ void buildGrid(range1f       *valueRanges,
                            const Gridlet *gridlets,
                            const float   *scalars,
                            const size_t   numGridlets,
                            const vec3i    maxGridletSize,
                            const vec3i    dims,
                            const box3f    worldBounds)
  {
    size_t gridletID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (gridletID >= numGridlets)
      return;

    const Gridlet &gridlet = gridlets[gridletID];

    int xyz = blockIdx.y * blockDim.y + threadIdx.y;
    int x = xyz%maxGridletSize.x;
    int y = xyz/maxGridletSize.x%maxGridletSize.y;
    int z = xyz/(maxGridletSize.x*maxGridletSize.y);
    int dimX = min(maxGridletSize.x,gridlet.dims.x);
    int dimY = min(maxGridletSize.y,gridlet.dims.y);
    int dimZ = min(maxGridletSize.z,gridlet.dims.z);

    if (x >= dimX || y >= dimY || z >= dimZ)
      return;

    const box3f &gridletBounds = gridlet.getBounds();

    const vec3f mcSize(worldBounds.size() / vec3f(dims));
    const vec3i loMC = projectOnGrid(gridletBounds.lower,dims,worldBounds);
    const vec3i upMC = projectOnGrid(gridletBounds.upper,dims,worldBounds);

    const vec3i numScalars = gridlet.dims+1;

    for (int mcz=loMC.z; mcz<=upMC.z; ++mcz) {
      for (int mcy=loMC.y; mcy<=upMC.y; ++mcy) {
        for (int mcx=loMC.x; mcx<=upMC.x; ++mcx) {
          const vec3i mcID(mcx,mcy,mcz);
          const box3f mcBounds(worldBounds.lower+vec3f(mcID)*mcSize,
                               worldBounds.lower+vec3f(mcID+1)*mcSize);


          range1f valueRange{+1e30f,-1e30f};
          /*for (int z=0; z<gridlet.dims.z; ++z)*/ {
            /*for (int y=0; y<gridlet.dims.y; ++y)*/ {
              /*for (int x=0; x<gridlet.dims.x; ++x)*/ {
                const float cellWidth = 1<<gridlet.level;
                const box3f cellBounds(vec3f((gridlet.lower+vec3i(x,y,z)) * cellWidth) + .5f*cellWidth,
                                       vec3f((gridlet.lower+vec3i(x+1,y+1,z+1)) * cellWidth) + .5f*cellWidth);
                if (mcBounds.overlaps(cellBounds)) {
                  vec3i imin(x,y,z);
                  vec3i imax(x+1,y+1,z+1);


                  float f1 = scalars[gridlet.begin+linearIndex(vec3i(imin.x,imin.y,imin.z),numScalars)];
                  float f2 = scalars[gridlet.begin+linearIndex(vec3i(imax.x,imin.y,imin.z),numScalars)];
                  float f3 = scalars[gridlet.begin+linearIndex(vec3i(imin.x,imax.y,imin.z),numScalars)];
                  float f4 = scalars[gridlet.begin+linearIndex(vec3i(imax.x,imax.y,imin.z),numScalars)];

                  float f5 = scalars[gridlet.begin+linearIndex(vec3i(imin.x,imin.y,imax.z),numScalars)];
                  float f6 = scalars[gridlet.begin+linearIndex(vec3i(imax.x,imin.y,imax.z),numScalars)];
                  float f7 = scalars[gridlet.begin+linearIndex(vec3i(imin.x,imax.y,imax.z),numScalars)];
                  float f8 = scalars[gridlet.begin+linearIndex(vec3i(imax.x,imax.y,imax.z),numScalars)];

                  if (!isnan(f1)) {
                    valueRange.lower = fminf(valueRange.lower,f1);
                    valueRange.upper = fmaxf(valueRange.upper,f1);
                  }

                  if (!isnan(f2)) {
                    valueRange.lower = fminf(valueRange.lower,f2);
                    valueRange.upper = fmaxf(valueRange.upper,f2);
                  }

                  if (!isnan(f3))  {
                    valueRange.lower = fminf(valueRange.lower,f3);
                    valueRange.upper = fmaxf(valueRange.upper,f3);
                  }

                  if (!isnan(f4)) {
                    valueRange.lower = fminf(valueRange.lower,f4);
                    valueRange.upper = fmaxf(valueRange.upper,f4);
                  }

                  if (!isnan(f5)) {
                    valueRange.lower = fminf(valueRange.lower,f5);
                    valueRange.upper = fmaxf(valueRange.upper,f5);
                  }

                  if (!isnan(f6)) {
                    valueRange.lower = fminf(valueRange.lower,f6);
                    valueRange.upper = fmaxf(valueRange.upper,f6);
                  }

                  if (!isnan(f7)) {
                    valueRange.lower = fminf(valueRange.lower,f7);
                    valueRange.upper = fmaxf(valueRange.upper,f7);
                  }

                  if (!isnan(f8)) {
                    valueRange.lower = fminf(valueRange.lower,f8);
                    valueRange.upper = fmaxf(valueRange.upper,f8);
                  }

                }
              }
            }
          }
          updateMC(mcID,dims,valueRange,valueRanges);
        }
      }
    }
  }

  // AMR cell overload
  __global__ void buildGrid(range1f       *valueRanges,
                            const AMRCell *cells,
                            const float   *scalars,
                            const size_t   numAmrCells,
                            const vec3i    dims,
                            const box3f    worldBounds)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= numAmrCells)
      return;

    const AMRCell &cell = cells[threadID];

    vec3i lower = cell.pos;
    vec3i upper = lower + (1<<cell.level);

    const vec3f halfCell = vec3f(1<<cell.level)*.5f;

    const vec3i loMC = projectOnGrid(vec3f(lower)-halfCell,dims,worldBounds);
    const vec3i upMC = projectOnGrid(vec3f(upper)+halfCell,dims,worldBounds);

    const float value = scalars[threadID];

    if (!isnan(value)) {
      for (int mcz=loMC.z; mcz<=upMC.z; ++mcz) {
        for (int mcy=loMC.y; mcy<=upMC.y; ++mcy) {
          for (int mcx=loMC.x; mcx<=upMC.x; ++mcx) {
            const vec3i mcID(mcx,mcy,mcz);
            updateMC(mcID,dims,value,valueRanges);
          }
        }
      }
    }
  }

  // ExaBrick overload
  __global__ void buildGrid(range1f      *valueRanges,
                            const ABR    *abrs,
                            const size_t  numABRs,
                            const vec3i   dims,
                            const box3f   worldBounds)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= numABRs)
      return;

    const box3f domain = abrs[threadID].domain;
    const range1f valueRange = abrs[threadID].valueRange;

    const vec3f mcSize(worldBounds.size() / vec3f(dims));
    const vec3i loMC = projectOnGrid(domain.lower,dims,worldBounds);
    const vec3i upMC = projectOnGrid(domain.upper,dims,worldBounds);

    for (int mcz=loMC.z; mcz<=upMC.z; ++mcz) {
      for (int mcy=loMC.y; mcy<=upMC.y; ++mcy) {
        for (int mcx=loMC.x; mcx<=upMC.x; ++mcx) {
          const vec3i mcID(mcx,mcy,mcz);
          updateMC(mcID,dims,valueRange,valueRanges);
        }
      }
    }
  }

  void Grid::build(OWLContext       owl,
                   AMRCellModel::SP model,
                   const owl::vec3i numMCs,
                   const owl::box3f bounds)
  {
    dims        = numMCs;
    worldBounds = bounds;

    valueRanges = owlDeviceBufferCreate(owl, OWL_USER_TYPE(range1f),
                                        dims.x*size_t(dims.y)*dims.z,
                                        nullptr);

    // Init with small floats
    {
      size_t numThreads = 1024;
      size_t numMCs = dims.x*size_t(dims.y)*dims.z;
      initGrid<<<iDivUp(numMCs, numThreads), numThreads>>>
        ((range1f *)owlBufferGetPointer(valueRanges,0),dims);

      // pre-allocating max-opacity buffer
      maxOpacities = owlDeviceBufferCreate(owl, OWL_FLOAT, numMCs, nullptr);
    }

    // Add contrib from AMR cells
    {
      size_t numThreads = 1024;
      size_t numAmrCells = owlBufferSizeInBytes(model->cellBuffer)/sizeof(AMRCell);
      std::cout << "DDA grid: adding " << numAmrCells << " AMR cells (non-dual!)\n";
      buildGrid<<<iDivUp(numAmrCells, numThreads), numThreads>>>(
        (range1f *)owlBufferGetPointer(valueRanges,0),
        (const AMRCell *)owlBufferGetPointer(model->cellBuffer,0),
        (const float *)owlBufferGetPointer(model->scalarBuffer,0),
        numAmrCells,dims,worldBounds);
      hipDeviceSynchronize();
      std::cout << hipGetErrorString(hipGetLastError()) << '\n';
    }
  }

  void Grid::build(OWLContext        owl,
                   ExaBrickModel::SP model,
                   const owl::vec3i  numMCs,
                   const owl::box3f  bounds)
  {
    dims        = numMCs;
    worldBounds = bounds;

    valueRanges = owlDeviceBufferCreate(owl, OWL_USER_TYPE(range1f),
                                        dims.x*size_t(dims.y)*dims.z,
                                        nullptr);

    // Init with small floats
    {
      size_t numThreads = 1024;
      size_t numMCs = dims.x*size_t(dims.y)*dims.z;
      initGrid<<<iDivUp(numMCs, numThreads), numThreads>>>
        ((range1f *)owlBufferGetPointer(valueRanges,0),dims);

      // pre-allocating max-opacity buffer
      maxOpacities = owlDeviceBufferCreate(owl, OWL_FLOAT, numMCs, nullptr);
    }

    // Add contrib from ExaBricks
    {
      double tfirst = getCurrentTime();
      size_t numThreads = 1024;
      size_t numABRs = owlBufferSizeInBytes(model->abrBuffer)/sizeof(ABR);
      std::cout << "DDA grid: adding " << numABRs << " ExaBrick ABRs\n";
      buildGrid<<<iDivUp(numABRs, numThreads), numThreads>>>(
        (range1f *)owlBufferGetPointer(valueRanges,0),
        (const ABR *)owlBufferGetPointer(model->abrBuffer,0),
        numABRs,dims,worldBounds);

      std::cout << hipGetErrorString(hipGetLastError()) << '\n';
      double tlast = getCurrentTime();
      std::cout << tlast-tfirst << '\n';
    }
  }

  void Grid::build(OWLContext         owl,
                   ExaStitchModel::SP model,
                   const owl::vec3i   numMCs,
                   const owl::box3f   bounds)
  {
    dims        = numMCs;
    worldBounds = bounds;

    valueRanges = owlDeviceBufferCreate(owl, OWL_USER_TYPE(range1f),
                                        dims.x*size_t(dims.y)*dims.z,
                                        nullptr);

    // Init with small floats
    {
      size_t numThreads = 1024;
      size_t numMCs = dims.x*size_t(dims.y)*dims.z;
      initGrid<<<iDivUp(numMCs, numThreads), numThreads>>>
        ((range1f *)owlBufferGetPointer(valueRanges,0),dims);

      // pre-allocating max-opacity buffer
      maxOpacities = owlDeviceBufferCreate(owl, OWL_FLOAT, numMCs, nullptr);
    }

    // Add contrib from uelems
    {
      size_t numThreads = 1024;
      size_t numElems = owlBufferSizeInBytes(model->indexBuffer)/sizeof(int[8]);
      std::cout << "DDA grid: adding " << numElems << " uelems\n";
      buildGrid<<<iDivUp(numElems, numThreads), numThreads>>>(
        (range1f *)owlBufferGetPointer(valueRanges,0),
        (const vec4f *)owlBufferGetPointer(model->vertexBuffer,0),
        (const int *)owlBufferGetPointer(model->indexBuffer,0),
        numElems,dims,worldBounds);
      hipDeviceSynchronize();
      std::cout << hipGetErrorString(hipGetLastError()) << '\n';
    }

    // Add contrib from gridlets
    {
      double tfirst = getCurrentTime();
      size_t numThreads = 1024;
      size_t numGridlets = owlBufferSizeInBytes(model->gridletBuffer)/sizeof(Gridlet);
      std::cout << "DDA grid: adding " << numGridlets << " gridlets\n";
      vec3i *maxGridletSize;
      hipMalloc(&maxGridletSize,sizeof(vec3i));
      vec3i init = 0;
      hipMemcpy(maxGridletSize,&init,sizeof(init),hipMemcpyHostToDevice);

      getMaxGridletSize<<<iDivUp(numGridlets, numThreads), numThreads>>>(
        (const Gridlet *)owlBufferGetPointer(model->gridletBuffer,0),
        numGridlets,maxGridletSize);

      vec3i hMaxGridletSize;
      hipMemcpy(&hMaxGridletSize,maxGridletSize,sizeof(hMaxGridletSize),
                 hipMemcpyDeviceToHost);

      dim3 gridDims(numGridlets,hMaxGridletSize.x*hMaxGridletSize.y*hMaxGridletSize.z);
      dim3 blockDims(64,16);
      dim3 numBlocks(iDivUp(gridDims.x,blockDims.x),
                     iDivUp(gridDims.y,blockDims.y));

      buildGrid<<<numBlocks, blockDims>>>(
        (range1f *)owlBufferGetPointer(valueRanges,0),
        (const Gridlet *)owlBufferGetPointer(model->gridletBuffer,0),
        (const float *)owlBufferGetPointer(model->gridletScalarBuffer,0),
        numGridlets,hMaxGridletSize,dims,worldBounds);

      hipFree(maxGridletSize);
      hipDeviceSynchronize();
      std::cout << hipGetErrorString(hipGetLastError()) << '\n';
      double tlast = getCurrentTime();
      std::cout << tlast-tfirst << '\n';
    }
  }

  bool Grid::initGPU(OWLContext owl, OWLModule module)
  {
#if EXA_STITCH_EXA_BRICK_TRAVERSAL_MODE == MC_BVH_TRAVERSAL
    // build BVH (tarversal method a)
    OWLVarDecl geomVars[]
    = {
       { "dims", OWL_INT3, OWL_OFFSETOF(MacroCellGeom,dims) },
       { "spacing", OWL_FLOAT3, OWL_OFFSETOF(MacroCellGeom,spacing) },
       { "origin", OWL_FLOAT3, OWL_OFFSETOF(MacroCellGeom,origin) },
       { nullptr /* sentinel to mark end of list */ }
    };

    const vec3f spacing(worldBounds.size() / vec3f(dims));

    geomType = owlGeomTypeCreate(owl, OWL_GEOM_USER, sizeof(MacroCellGeom), geomVars, -1);
    owlGeomTypeSetBoundsProg   (geomType, module, "MacroCellGeomBounds");
    owlGeomTypeSetIntersectProg(geomType, RADIANCE_RAY_TYPE, module, "MacroCellGeomIsect");
    owlGeomTypeSetClosestHit   (geomType, RADIANCE_RAY_TYPE, module, "MacroCellGeomCH");
    OWLGeom geom = owlGeomCreate(owl, geomType);
    owlGeomSetPrimCount(geom, size_t(dims.x)*size_t(dims.y)*size_t(dims.z));
    owlGeomSet3i(geom,"dims", dims.x, dims.y, dims.z);
    owlGeomSet3f(geom,"spacing", spacing.x, spacing.y, spacing.z);
    owlGeomSet3f(geom,"origin", worldBounds.lower.x, worldBounds.lower.y, worldBounds.lower.z);

    owlBuildPrograms(owl);

    blas = owlUserGeomGroupCreate(owl, 1, &geom);
    owlGroupBuildAccel(blas);
    tlas = owlInstanceGroupCreate(owl, 1);
    owlInstanceGroupSetChild(tlas, 0, blas);
    owlGroupBuildAccel(tlas);
#endif

    // init device traversable for DDA (traversal method b)
    deviceTraversable.dims = dims;
    deviceTraversable.bounds = worldBounds;

    return true;
  }

  __global__ void computeMaxOpacitiesGPU(float         *maxOpacities,
                                         const range1f *valueRanges,
                                         const vec4f   *colorMap,
                                         size_t         numMCs,
                                         size_t         numColors,
                                         range1f        xfRange)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;

    if (threadID >= numMCs)
      return;

    range1f valueRange = valueRanges[threadID];

    if (valueRange.upper < valueRange.lower) {
      maxOpacities[threadID] = 0.f;
      return;
    }

    valueRange.lower -= xfRange.lower;
    valueRange.lower /= xfRange.upper-xfRange.lower;
    valueRange.upper -= xfRange.lower;
    valueRange.upper /= xfRange.upper-xfRange.lower;

    int lo = clamp(int(valueRange.lower*(numColors-1)),0,(int)numColors-1);
    int hi = clamp(int(valueRange.upper*(numColors-1))+1,0,(int)numColors-1);

    float maxOpacity = 0.f;
    for (int i=lo; i<=hi; ++i) {
      maxOpacity = fmaxf(maxOpacity,colorMap[i].w);
    }
    maxOpacities[threadID] = maxOpacity;
  }

  void Grid::computeMaxOpacities(OWLContext owl, OWLBuffer colorMap, range1f xfRange)
  {
    size_t numMCs = dims.x*size_t(dims.y)*dims.z;
    size_t numColors = owlBufferSizeInBytes(colorMap)/sizeof(vec4f);

    size_t numThreads = 1024;
    computeMaxOpacitiesGPU<<<iDivUp(numMCs, numThreads), numThreads>>>(
      (float *)owlBufferGetPointer(maxOpacities,0),
      (const range1f *)owlBufferGetPointer(valueRanges,0),
      (const vec4f *)owlBufferGetPointer(colorMap,0),
      numMCs,numColors,xfRange);
  }

} // ::exa

// vim: sw=2:expandtab:softtabstop=2:ts=2:cino=\:0g0t0

