#include "hip/hip_runtime.h"
#include <vector>
#include "VolumeLines.h"
#include "atomicOp.cuh"
#include "hilbert.h"

inline int64_t __host__ __device__ iDivUp(int64_t a, int64_t b)
{
  return (a + b - 1) / b;
}

__global__ void fillGPU(hipSurfaceObject_t surfaceObj, int w, int h)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= w || y >= h)
    return;

  if ((x/16) % 2 == (y/16) % 2)
    surf2Dwrite(make_float4(1.f,1.f,1.f,1.f), surfaceObj, x * sizeof(float4), h-y-1);
  else
    surf2Dwrite(make_float4(.9f,.9f,.9f,.9f), surfaceObj, x * sizeof(float4), h-y-1);
}

__global__ void renderGPU(hipSurfaceObject_t surfaceObj, float *grid, int w, int h)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (x >= w)
    return;

  int H=owl::clamp(int(grid[x]*h),0,h-1);
  for (int y=0; y<=H; ++y) {
    float4 src;
    surf2Dread(&src, surfaceObj, x * sizeof(float4), h-y-1);
    owl::vec3f c(0.f);
    float4 color = make_float4(src.x*0.5f+c.x*0.5f,
                               src.y*0.5f+c.y*0.5f,
                               src.z*0.5f+c.z*0.5f,
                               1.f);
    surf2Dwrite(color, surfaceObj, x * sizeof(float4), h-y-1);
  }
}

inline __device__
vec4f lookupTransferFunction(float f,
                             const vec4f *colorMap,
                             const int numColors,
                             const range1f xfDomain)
{
  if (xfDomain.lower >= xfDomain.upper)
    return vec4f(0.f);

  f -= xfDomain.lower;
  f /= (xfDomain.upper-xfDomain.lower);
  if (numColors == 0)
    return vec4f(0.f);

  f = max(0.f,min(1.f,f));
  int i = min(numColors-1,int(f * numColors));
  return colorMap[i];
}

__global__ void basisRasterCells(float *grid,
                                 float *weights,
                                 int dims,
                                 const exa::VolumeLines::Cell *cells,
                                 int numCells,
                                 range1f cellBounds,
                                 const vec4f *colorMap,
                                 const int numColors,
                                 const range1f xfDomain)
{
  int primID = blockIdx.x * blockDim.x + threadIdx.x;

  if (primID >= numCells)
    return;

  const auto &cell = cells[primID];
  range1f bounds = cell.getBounds();
  float p1 = bounds.lower;
  float p2 = bounds.upper;

  // Project onto grid (TODO: move to function..)
  float x1_01 = (p1-cellBounds.lower)/(cellBounds.upper-cellBounds.lower);
  float x2_01 = (p2-cellBounds.lower)/(cellBounds.upper-cellBounds.lower);

  int x1 = owl::clamp(int(x1_01*float(dims)),0,dims-1);
  int x2 = owl::clamp(int(x2_01*float(dims)),0,dims-1);

  for (int x=x1; x<=x2; ++x) {
    // TODO: that's a box-shaped basis function
    // this _might_ be ok, but only if we have
    // many cells..
    const vec4f color = lookupTransferFunction(cell.value,colorMap,numColors,xfDomain);
    atomicAdd(&grid[x], color.w);
    atomicAdd(&weights[x], 1.f);
  }
}

__global__ void basisAverageGridCells(float *grid,
                                      float *weights,
                                      int dims)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x >= dims)
    return;

  //if (weights[x] > 0.f)
  //  grid[x] /= weights[x];
}

namespace exa {
  VolumeLines::VolumeLines()
  {
    // cells.resize(2);
    // for (size_t i=0; i<cells.size(); ++i) {
    //   std::vector<Cell> cs(2000000);
    //   for (size_t j=0; j<cs.size(); ++j) {
    //     int level=0;
    //     //float value(cs.size()/2);
    //     float value = rand()/float(RAND_MAX);
    //     cs[j] = {(int)j,value,level};
    //     cellBounds.extend(cs[j].getBounds());
    //   }

    //   hipMalloc(&cells[i], cs.size()*sizeof(cs[0]));
    //   hipMemcpy(cells[i], cs.data(), cs.size()*sizeof(cs[0]), hipMemcpyHostToDevice);

    //   numCells = cs.size();
    // }
  }

  VolumeLines::~VolumeLines()
  {
    for (size_t i=0; i<cells.size(); ++i) {
      hipFree(cells[i]);
    }

    for (size_t i=0; i<grids1D.size(); ++i) {
      hipFree(grids1D[i]);
    }
  }

  void VolumeLines::reset(const ExaBrickModel::SP &model)
  {
    for (size_t i=0; i<cells.size(); ++i) {
      hipFree(cells[i]);
    }

    cells.resize(1);
    cellBounds = {};
    std::vector<Cell> cs;
    for (size_t i=0; i<model->bricks.size(); ++i) {
      const ExaBrick &brick = model->bricks[i];
      for (int z=0; z<brick.size.z; ++z) {
        for (int y=0; y<brick.size.y; ++y) {
          for (int x=0; x<brick.size.x; ++x) {
            int idx = brick.getIndexIndex({x,y,z});
            range1f vr = model->valueRange;
            #if 1
            float val = model->scalars[idx];
            #else
            float val = (model->scalars[idx]-vr.lower)/(vr.upper-vr.lower);
            #endif
            Cell c{idx,val,brick.level};
            cs.push_back(c);
            cellBounds.extend(c.getBounds());
          }
        }
      }
    }

    for (Cell &c : cs) {
      vec3i centroid = c.getBounds().center();
      vec3f centroid01(centroid);
      centroid01 = (centroid01-vec3f(cellBounds.lower)) / vec3f(cellBounds.upper-cellBounds.lower);
      vec3f quantized(centroid01);
      quantized *= float(1<<16);
      const bitmask_t coord[3] = {
        bitmask_t(quantized.x),
        bitmask_t(quantized.y),
        bitmask_t(quantized.z)
      };
      c.hilbertID = hilbert_c2i(3, 16, coord);
    }

    std::sort(cs.begin(),cs.end(),
              [](const Cell &a, const Cell &b)
              { return a.hilbertID < b.hilbertID; }
            );

    hipMalloc(&cells[0], cs.size()*sizeof(cs[0]));
    hipMemcpy(cells[0], cs.data(), cs.size()*sizeof(cs[0]), hipMemcpyHostToDevice);
    numCells = cs.size();
    updated_ = true;
  }

  void VolumeLines::draw(hipSurfaceObject_t surfaceObj, int w, int h)
  {
    // Fill background
    {
      dim3 blockSize;
      blockSize.x = 16;
      blockSize.y = 16;
      
      dim3 gridSize;
      gridSize.x = iDivUp(w,blockSize.x);
      gridSize.y = iDivUp(w,blockSize.y);

      fillGPU<<<gridSize, blockSize>>>(surfaceObj,w,h);
    }

    if (updated_ && xf.deviceColorMap) {
      for (size_t i=0; i<grids1D.size(); ++i) {
        hipFree(grids1D[i]);
      }

      grids1D.clear();

      // raster cells onto 1D grids
      for (size_t i=0; i<cells.size(); ++i) {
        float *grid;
        hipMalloc(&grid, sizeof(float)*w);
        hipMemset(grid, 0, sizeof(float)*w);

        float *weights;
        hipMalloc(&weights, sizeof(float)*w);
        hipMemset(weights, 0, sizeof(float)*w);

        // TODO: set per channel!
        range1f r{
         xf.absDomain.lower + (xf.relDomain.lower/100.f) * (xf.absDomain.upper-xf.absDomain.lower),
         xf.absDomain.lower + (xf.relDomain.upper/100.f) * (xf.absDomain.upper-xf.absDomain.lower)
        };

        size_t numThreads = 1024;
        basisRasterCells<<<iDivUp(numCells,numThreads),numThreads>>>(
          grid, weights, w, cells[i], numCells, cellBounds,
          xf.deviceColorMap, xf.colorMap.size(), r);

        grids1D.push_back(grid);

        basisAverageGridCells<<<iDivUp(w,numThreads),numThreads>>>(
          grid, weights, w);

        hipFree(weights);
      }

      updated_ = false;
    }

    // render to texture
    for (size_t i=0; i<grids1D.size(); ++i) {
      size_t numThreads = 1024;
      renderGPU<<<iDivUp(w,numThreads),numThreads>>>(
        surfaceObj,grids1D[i],w,h);
    }
  }

  void VolumeLines::setColorMap(const std::vector<vec4f> &newCM)
  {
    xf.colorMap = newCM;

    hipFree(xf.deviceColorMap);
    hipMalloc(&xf.deviceColorMap, newCM.size()*sizeof(newCM[0]));
    hipMemcpy(xf.deviceColorMap, newCM.data(), newCM.size()*sizeof(newCM[0]),
               hipMemcpyHostToDevice);

    updated_ = true;
  }

  void VolumeLines::setRange(interval<float> xfDomain)
  {
    xf.absDomain = xfDomain;
    updated_ = true;
  }

  void VolumeLines::setRelDomain(interval<float> relDomain)
  {
    xf.relDomain = relDomain;
    updated_ = true;
  }

  void VolumeLines::setOpacityScale(float scale)
  {
    xf.opacityScale = scale;
    updated_ = true;
  }
} // ::exa
// vim: sw=2:expandtab:softtabstop=2:ts=2:cino=\:0g0t0
