#include "hip/hip_runtime.h"
#include <vector>
#include <hipcub/hipcub.hpp>
#include "VolumeLines.h"
#include "atomicOp.cuh"
#include "hilbert.h"

inline int64_t __host__ __device__ iDivUp(int64_t a, int64_t b)
{
  return (a + b - 1) / b;
}

// CustomMin functor
struct CustomMin
{
  template <typename T>
    HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
    T operator()(const T &a, const T &b) const {
      return (b < a) ? b : a;
    }
};

__global__ void fillGPU(hipSurfaceObject_t surfaceObj, int w, int h)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= w || y >= h)
    return;

  if ((x/16) % 2 == (y/16) % 2)
    surf2Dwrite(make_float4(.4f,.4f,.4f,1.f), surfaceObj, x * sizeof(float4), h-y-1);
  else
    surf2Dwrite(make_float4(.3f,.3f,.3f,1.f), surfaceObj, x * sizeof(float4), h-y-1);
}

__global__ void renderGPU(hipSurfaceObject_t surfaceObj,
                          exa::VolumeLines::GridCell *grid, int w, int h)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (x >= w)
    return;

  int H=owl::clamp(int(grid[x].color.w*h),0,h-1);
  for (int y=0; y<=H; ++y) {
    float4 src;
    surf2Dread(&src, surfaceObj, x * sizeof(float4), h-y-1);
    owl::vec3f c(grid[x].color);
    float4 color = make_float4(src.x*0.f+c.x*1.f,
                               src.y*0.f+c.y*1.f,
                               src.z*0.f+c.z*1.f,
                               1.f);
    surf2Dwrite(color, surfaceObj, x * sizeof(float4), h-y-1);
  }
}

inline __device__
vec4f lookupTransferFunction(float f,
                             const vec4f *colorMap,
                             const int numColors,
                             const range1f xfDomain)
{
  if (xfDomain.lower >= xfDomain.upper)
    return vec4f(0.f);

  f -= xfDomain.lower;
  f /= (xfDomain.upper-xfDomain.lower);
  if (numColors == 0)
    return vec4f(0.f);

  f = max(0.f,min(1.f,f));
  int i = min(numColors-1,int(f * numColors));
  return colorMap[i];
}

__global__ void assignImportance(exa::VolumeLines::Cell *cells,
                                 float *importance,
                                 int numCells,
                                 // for the moment use TF to assign importance..
                                 const vec4f *colorMap,
                                 const int numColors,
                                 const range1f xfDomain,
                                 float P)
{
  int primID = blockIdx.x * blockDim.x + threadIdx.x;

  if (primID >= numCells)
    return;

  vec4f color = lookupTransferFunction(cells[primID].value,colorMap,numColors,xfDomain);
  // this will later become the diff of two time steps
  importance[primID] = fmaxf(0.025f, powf(color.w,P));
}

__global__ void basisRasterCells(exa::VolumeLines::GridCell *grid,
                                 float *weights,
                                 int dims,
                                 const exa::VolumeLines::Cell *cells,
                                 const float *cumulativeImportance,
                                 int numCells,
                                 range1f cellBounds)
{
  int primID = blockIdx.x * blockDim.x + threadIdx.x;

  if (primID >= numCells)
    return;

  const auto &cell = cells[primID];
  range1f bounds = cell.getBounds();
  float p1 = bounds.lower;
  float p2 = bounds.upper;

  // Project onto grid (TODO: move to function..)
  float x1_01 = (p1-cellBounds.lower)/(cellBounds.upper-cellBounds.lower);
  float x2_01 = (p2-cellBounds.lower)/(cellBounds.upper-cellBounds.lower);

  int x1 = owl::clamp(int(x1_01*float(dims)),0,dims-1);
  int x2 = owl::clamp(int(x2_01*float(dims)),0,dims-1);

  //float importanceScale = cumulativeImportance[primID]/cumulativeImportance[numCells-1];

  for (int x=x1; x<=x2; ++x) {
    int X = x;//*importanceScale;
    // TODO: that's a box-shaped basis function
    // this _might_ be ok, but only if we have
    // many cells..
    atomicAdd(&grid[X].value, cell.value);
    atomicAdd(&weights[X], 1.f);
  }
}

__global__ void basisAverageGridCells(exa::VolumeLines::GridCell *grid,
                                      float *weights,
                                      int dims)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x >= dims)
    return;

  if (weights[x] > 0.f)
    grid[x].value /= weights[x];
}

__global__ void postClassifyCells(exa::VolumeLines::GridCell *grid,
                                  int dims,
                                  const vec4f *colorMap,
                                  const int numColors,
                                  const range1f xfDomain)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x >= dims)
    return;

  grid[x].color = lookupTransferFunction(grid[x].value,colorMap,numColors,xfDomain);
}

namespace exa {
  VolumeLines::VolumeLines()
  {
    // cells.resize(2);
    // for (size_t i=0; i<cells.size(); ++i) {
    //   std::vector<Cell> cs(2000000);
    //   for (size_t j=0; j<cs.size(); ++j) {
    //     int level=0;
    //     //float value(cs.size()/2);
    //     float value = rand()/float(RAND_MAX);
    //     cs[j] = {(int)j,value,level};
    //     cellBounds.extend(cs[j].getBounds());
    //   }

    //   hipMalloc(&cells[i], cs.size()*sizeof(cs[0]));
    //   hipMemcpy(cells[i], cs.data(), cs.size()*sizeof(cs[0]), hipMemcpyHostToDevice);

    //   numCells = cs.size();
    // }
  }

  VolumeLines::~VolumeLines()
  {
    for (size_t i=0; i<cells.size(); ++i) {
      hipFree(cells[i]);
    }

    for (size_t i=0; i<grids1D.size(); ++i) {
      hipFree(grids1D[i]);
    }
  }

  void VolumeLines::reset(const ExaBrickModel::SP &model)
  {
    for (size_t i=0; i<cells.size(); ++i) {
      hipFree(cells[i]);
    }

    cells.resize(1);
    cellBounds = {};
    centroidBounds = {};
    std::vector<Cell> cs;
    for (size_t i=0; i<model->bricks.size(); ++i) {
      const ExaBrick &brick = model->bricks[i];
      for (int z=0; z<brick.size.z; ++z) {
        for (int y=0; y<brick.size.y; ++y) {
          for (int x=0; x<brick.size.x; ++x) {
            int idx = brick.getIndexIndex({x,y,z});
            range1f vr = model->valueRange;
            #if 1
            float val = model->scalars[idx];
            #else
            float val = (model->scalars[idx]-vr.lower)/(vr.upper-vr.lower);
            #endif
            int lower = cs.empty() ? 0 : cs.back().getBounds().upper;
            Cell c{lower,val,brick.level};
            cs.push_back(c);
            centroidBounds.extend(c.getBounds().center());
            cellBounds.extend(c.getBounds());
          }
        }
      }
    }

    #pragma omp parallel for
    for (size_t i=0; i<cs.size(); ++i) {
      Cell &c = cs[i];
      vec3i centroid = c.getBounds().center();
      vec3f centroid01(centroid);
      centroid01 = (centroid01-vec3f(centroidBounds.lower)) / vec3f(centroidBounds.upper-centroidBounds.lower);

      vec3f quantized(centroid01);
      quantized *= float(1<<16);
      const bitmask_t coord[3] = {
        bitmask_t(quantized.x),
        bitmask_t(quantized.y),
        bitmask_t(quantized.z)
      };
      c.hilbertID = hilbert_c2i(3, 16, coord);
    }

    std::sort(cs.begin(),cs.end(),
              [](const Cell &a, const Cell &b)
              { return a.hilbertID < b.hilbertID; }
            );

    hipMalloc(&cells[0], cs.size()*sizeof(cs[0]));
    hipMemcpy(cells[0], cs.data(), cs.size()*sizeof(cs[0]), hipMemcpyHostToDevice);
    numCells = cs.size();
    updated_ = true;
  }

  void VolumeLines::draw(hipSurfaceObject_t surfaceObj, int w, int h)
  {
    // Fill background
    {
      dim3 blockSize;
      blockSize.x = 16;
      blockSize.y = 16;
      
      dim3 gridSize;
      gridSize.x = iDivUp(w,blockSize.x);
      gridSize.y = iDivUp(w,blockSize.y);

      fillGPU<<<gridSize, blockSize>>>(surfaceObj,w,h);
    }

    if (updated_ && xf.deviceColorMap) {
      for (size_t i=0; i<grids1D.size(); ++i) {
        hipFree(grids1D[i]);
      }

      grids1D.clear();

      // raster cells onto 1D grids
      for (size_t i=0; i<cells.size(); ++i) {
        GridCell *grid;
        hipMalloc(&grid, sizeof(GridCell)*w);
        hipMemset(grid, 0, sizeof(GridCell)*w);

        float *weights;
        hipMalloc(&weights, sizeof(float)*w);
        hipMemset(weights, 0, sizeof(float)*w);

        // TODO: set per channel!
        range1f r{
         xf.absDomain.lower + (xf.relDomain.lower/100.f) * (xf.absDomain.upper-xf.absDomain.lower),
         xf.absDomain.lower + (xf.relDomain.upper/100.f) * (xf.absDomain.upper-xf.absDomain.lower)
        };

        size_t numThreads = 1024;

        float P=1.f; // TODO: user param
        float *importance;
        hipMalloc(&importance, sizeof(float)*numCells);
        assignImportance<<<iDivUp(numCells,numThreads),numThreads>>>(
          cells[i], importance, numCells, xf.deviceColorMap, xf.colorMap.size(), r, P);

        // TODO: alloc once (all these arrays!!)
        void *tempStorage = nullptr;
        size_t tempStorageBytes = 0;
        float *cumulativeImportance;
        hipMalloc(&cumulativeImportance, sizeof(float)*numCells);
        CustomMin min_op;
        hipcub::DeviceScan::ExclusiveSum(tempStorage, tempStorageBytes, importance,
                                      cumulativeImportance, numCells);
        hipMalloc(&tempStorage, tempStorageBytes);
        hipcub::DeviceScan::ExclusiveSum(tempStorage, tempStorageBytes, importance,
                                      cumulativeImportance, numCells);
        hipFree(importance);
        hipFree(tempStorage);
        //std::vector<float> cumImp(numCells);
        //hipMemcpy(cumImp.data(),cumulativeImportance,sizeof(float)*numCells,hipMemcpyDeviceToHost);
        //for (size_t j=0; j<numCells; ++j) {
        //  std::cout << cumImp[j] << '\n';
        //}

        basisRasterCells<<<iDivUp(numCells,numThreads),numThreads>>>(
          grid, weights, w, cells[i], cumulativeImportance, numCells, cellBounds);

        hipFree(cumulativeImportance);

        grids1D.push_back(grid);

        basisAverageGridCells<<<iDivUp(w,numThreads),numThreads>>>(
          grid, weights, w);

        hipFree(weights);

        postClassifyCells<<<iDivUp(numCells,numThreads),numThreads>>>(
          grid, w, xf.deviceColorMap, xf.colorMap.size(), r);

      }

      updated_ = false;
    }

    // render to texture
    for (size_t i=0; i<grids1D.size(); ++i) {
      size_t numThreads = 1024;
      renderGPU<<<iDivUp(w,numThreads),numThreads>>>(
        surfaceObj,grids1D[i],w,h);
    }
  }

  void VolumeLines::setColorMap(const std::vector<vec4f> &newCM)
  {
    xf.colorMap = newCM;

    hipFree(xf.deviceColorMap);
    hipMalloc(&xf.deviceColorMap, newCM.size()*sizeof(newCM[0]));
    hipMemcpy(xf.deviceColorMap, newCM.data(), newCM.size()*sizeof(newCM[0]),
               hipMemcpyHostToDevice);

    updated_ = true;
  }

  void VolumeLines::setRange(interval<float> xfDomain)
  {
    xf.absDomain = xfDomain;
    updated_ = true;
  }

  void VolumeLines::setRelDomain(interval<float> relDomain)
  {
    xf.relDomain = relDomain;
    updated_ = true;
  }

  void VolumeLines::setOpacityScale(float scale)
  {
    xf.opacityScale = scale;
    updated_ = true;
  }
} // ::exa
// vim: sw=2:expandtab:softtabstop=2:ts=2:cino=\:0g0t0
