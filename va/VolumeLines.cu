#include "hip/hip_runtime.h"
#include <vector>
#include "VolumeLines.h"
#include "atomicOp.cuh"
#include "hilbert.h"

inline int64_t __host__ __device__ iDivUp(int64_t a, int64_t b)
{
  return (a + b - 1) / b;
}

__global__ void fillGPU(hipSurfaceObject_t surfaceObj, int w, int h)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= w || y >= h)
    return;

  if ((x/16) % 2 == (y/16) % 2)
    surf2Dwrite(make_float4(.4f,.4f,.4f,1.f), surfaceObj, x * sizeof(float4), h-y-1);
  else
    surf2Dwrite(make_float4(.3f,.3f,.3f,1.f), surfaceObj, x * sizeof(float4), h-y-1);
}

__global__ void renderGPU(hipSurfaceObject_t surfaceObj,
                          exa::VolumeLines::GridCell *grid, int w, int h)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (x >= w)
    return;

  int H=owl::clamp(int(grid[x].color.w*h),0,h-1);
  for (int y=0; y<=H; ++y) {
    float4 src;
    surf2Dread(&src, surfaceObj, x * sizeof(float4), h-y-1);
    owl::vec3f c(grid[x].color);
    float4 color = make_float4(src.x*0.f+c.x*1.f,
                               src.y*0.f+c.y*1.f,
                               src.z*0.f+c.z*1.f,
                               1.f);
    surf2Dwrite(color, surfaceObj, x * sizeof(float4), h-y-1);
  }
}

inline __device__
vec4f lookupTransferFunction(float f,
                             const vec4f *colorMap,
                             const int numColors,
                             const range1f xfDomain)
{
  if (xfDomain.lower >= xfDomain.upper)
    return vec4f(0.f);

  f -= xfDomain.lower;
  f /= (xfDomain.upper-xfDomain.lower);
  if (numColors == 0)
    return vec4f(0.f);

  f = max(0.f,min(1.f,f));
  int i = min(numColors-1,int(f * numColors));
  return colorMap[i];
}

__global__ void basisRasterCells(exa::VolumeLines::GridCell *grid,
                                 float *weights,
                                 int dims,
                                 const exa::VolumeLines::Cell *cells,
                                 int numCells,
                                 range1f cellBounds)
{
  int primID = blockIdx.x * blockDim.x + threadIdx.x;

  if (primID >= numCells)
    return;

  const auto &cell = cells[primID];
  range1f bounds = cell.getBounds();
  float p1 = bounds.lower;
  float p2 = bounds.upper;

  // Project onto grid (TODO: move to function..)
  float x1_01 = (p1-cellBounds.lower)/(cellBounds.upper-cellBounds.lower);
  float x2_01 = (p2-cellBounds.lower)/(cellBounds.upper-cellBounds.lower);

  int x1 = owl::clamp(int(x1_01*float(dims)),0,dims-1);
  int x2 = owl::clamp(int(x2_01*float(dims)),0,dims-1);

  for (int x=x1; x<=x2; ++x) {
    // TODO: that's a box-shaped basis function
    // this _might_ be ok, but only if we have
    // many cells..
    atomicAdd(&grid[x].value, cell.value);
    atomicAdd(&weights[x], 1.f);
  }
}

__global__ void basisAverageGridCells(exa::VolumeLines::GridCell *grid,
                                      float *weights,
                                      int dims)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x >= dims)
    return;

  if (weights[x] > 0.f)
    grid[x].value /= weights[x];
}

__global__ void postClassifyCells(exa::VolumeLines::GridCell *grid,
                                  int dims,
                                  const vec4f *colorMap,
                                  const int numColors,
                                  const range1f xfDomain)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x >= dims)
    return;

  grid[x].color = lookupTransferFunction(grid[x].value,colorMap,numColors,xfDomain);
}

namespace exa {
  VolumeLines::VolumeLines()
  {
    // cells.resize(2);
    // for (size_t i=0; i<cells.size(); ++i) {
    //   std::vector<Cell> cs(2000000);
    //   for (size_t j=0; j<cs.size(); ++j) {
    //     int level=0;
    //     //float value(cs.size()/2);
    //     float value = rand()/float(RAND_MAX);
    //     cs[j] = {(int)j,value,level};
    //     cellBounds.extend(cs[j].getBounds());
    //   }

    //   hipMalloc(&cells[i], cs.size()*sizeof(cs[0]));
    //   hipMemcpy(cells[i], cs.data(), cs.size()*sizeof(cs[0]), hipMemcpyHostToDevice);

    //   numCells = cs.size();
    // }
  }

  VolumeLines::~VolumeLines()
  {
    for (size_t i=0; i<cells.size(); ++i) {
      hipFree(cells[i]);
    }

    for (size_t i=0; i<grids1D.size(); ++i) {
      hipFree(grids1D[i]);
    }
  }

  void VolumeLines::reset(const ExaBrickModel::SP &model)
  {
    for (size_t i=0; i<cells.size(); ++i) {
      hipFree(cells[i]);
    }

    cells.resize(1);
    cellBounds = {};
    centroidBounds = {};
    std::vector<Cell> cs;
    for (size_t i=0; i<model->bricks.size(); ++i) {
      const ExaBrick &brick = model->bricks[i];
      for (int z=0; z<brick.size.z; ++z) {
        for (int y=0; y<brick.size.y; ++y) {
          for (int x=0; x<brick.size.x; ++x) {
            int idx = brick.getIndexIndex({x,y,z});
            range1f vr = model->valueRange;
            #if 1
            float val = model->scalars[idx];
            #else
            float val = (model->scalars[idx]-vr.lower)/(vr.upper-vr.lower);
            #endif
            Cell c{idx,val,brick.level};
            cs.push_back(c);
            centroidBounds.extend(c.getBounds().center());
            cellBounds.extend(c.getBounds());
          }
        }
      }
    }

    #pragma omp parallel for
    for (size_t i=0; i<cs.size(); ++i) {
      Cell &c = cs[i];
      vec3i centroid = c.getBounds().center();
      vec3f centroid01(centroid);
      centroid01 = (centroid01-vec3f(centroidBounds.lower)) / vec3f(centroidBounds.upper-centroidBounds.lower);

      vec3f quantized(centroid01);
      quantized *= float(1<<16);
      const bitmask_t coord[3] = {
        bitmask_t(quantized.x),
        bitmask_t(quantized.y),
        bitmask_t(quantized.z)
      };
      c.hilbertID = hilbert_c2i(3, 16, coord);
    }

    std::sort(cs.begin(),cs.end(),
              [](const Cell &a, const Cell &b)
              { return a.hilbertID < b.hilbertID; }
            );

    hipMalloc(&cells[0], cs.size()*sizeof(cs[0]));
    hipMemcpy(cells[0], cs.data(), cs.size()*sizeof(cs[0]), hipMemcpyHostToDevice);
    numCells = cs.size();
    updated_ = true;
  }

  void VolumeLines::draw(hipSurfaceObject_t surfaceObj, int w, int h)
  {
    // Fill background
    {
      dim3 blockSize;
      blockSize.x = 16;
      blockSize.y = 16;
      
      dim3 gridSize;
      gridSize.x = iDivUp(w,blockSize.x);
      gridSize.y = iDivUp(w,blockSize.y);

      fillGPU<<<gridSize, blockSize>>>(surfaceObj,w,h);
    }

    if (updated_ && xf.deviceColorMap) {
      for (size_t i=0; i<grids1D.size(); ++i) {
        hipFree(grids1D[i]);
      }

      grids1D.clear();

      // raster cells onto 1D grids
      for (size_t i=0; i<cells.size(); ++i) {
        GridCell *grid;
        hipMalloc(&grid, sizeof(GridCell)*w);
        hipMemset(grid, 0, sizeof(GridCell)*w);

        float *weights;
        hipMalloc(&weights, sizeof(float)*w);
        hipMemset(weights, 0, sizeof(float)*w);

        // TODO: set per channel!
        range1f r{
         xf.absDomain.lower + (xf.relDomain.lower/100.f) * (xf.absDomain.upper-xf.absDomain.lower),
         xf.absDomain.lower + (xf.relDomain.upper/100.f) * (xf.absDomain.upper-xf.absDomain.lower)
        };

        size_t numThreads = 1024;
        basisRasterCells<<<iDivUp(numCells,numThreads),numThreads>>>(
          grid, weights, w, cells[i], numCells, cellBounds);

        grids1D.push_back(grid);

        basisAverageGridCells<<<iDivUp(w,numThreads),numThreads>>>(
          grid, weights, w);

        hipFree(weights);

        postClassifyCells<<<iDivUp(numCells,numThreads),numThreads>>>(
          grid, w, xf.deviceColorMap, xf.colorMap.size(), r);

      }

      updated_ = false;
    }

    // render to texture
    for (size_t i=0; i<grids1D.size(); ++i) {
      size_t numThreads = 1024;
      renderGPU<<<iDivUp(w,numThreads),numThreads>>>(
        surfaceObj,grids1D[i],w,h);
    }
  }

  void VolumeLines::setColorMap(const std::vector<vec4f> &newCM)
  {
    xf.colorMap = newCM;

    hipFree(xf.deviceColorMap);
    hipMalloc(&xf.deviceColorMap, newCM.size()*sizeof(newCM[0]));
    hipMemcpy(xf.deviceColorMap, newCM.data(), newCM.size()*sizeof(newCM[0]),
               hipMemcpyHostToDevice);

    updated_ = true;
  }

  void VolumeLines::setRange(interval<float> xfDomain)
  {
    xf.absDomain = xfDomain;
    updated_ = true;
  }

  void VolumeLines::setRelDomain(interval<float> relDomain)
  {
    xf.relDomain = relDomain;
    updated_ = true;
  }

  void VolumeLines::setOpacityScale(float scale)
  {
    xf.opacityScale = scale;
    updated_ = true;
  }
} // ::exa
// vim: sw=2:expandtab:softtabstop=2:ts=2:cino=\:0g0t0
