#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2022-2022 Stefan Zellmann                                      //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "ExaBrickModel.h"
#include "atomicOp.cuh"

namespace exa {

  inline int64_t __both__ iDivUp(int64_t a, int64_t b)
  {
    return (a + b - 1) / b;
  }

  __global__ void computeMaxOpacitiesForBricks(float         *exaBrickMaxOpacities,
                                               const range1f *brickValueRanges,
                                               const vec4f   *colorMap,
                                               size_t         numBricks,
                                               size_t         numColors,
                                               range1f        xfRange)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;
    if (threadID >= numBricks)
      return;

    range1f valueRange = brickValueRanges[threadID];

    if (valueRange.upper < valueRange.lower) {
      exaBrickMaxOpacities[threadID] = 0.f;
      return;
    }

    valueRange.lower -= xfRange.lower;
    valueRange.lower /= xfRange.upper-xfRange.lower;
    valueRange.upper -= xfRange.lower;
    valueRange.upper /= xfRange.upper-xfRange.lower;

    int lo = clamp(int(valueRange.lower*(numColors-1)),  0,(int)numColors-1);
    int hi = clamp(int(valueRange.upper*(numColors-1))+1,0,(int)numColors-1);

    float maxOpacity = 0.f;
    for (int i=lo; i<=hi; ++i) {
      maxOpacity = fmaxf(maxOpacity,colorMap[i].w);
    }

    exaBrickMaxOpacities[threadID] = maxOpacity;
  }

  __global__ void computeMaxOpacitiesForABRs(float       *abrMaxOpacities,
                                             const ABR   *abrs,
                                             const vec4f *colorMap,
                                             size_t       numABRs,
                                             size_t       numColors,
                                             range1f      xfRange)
  {
    size_t threadID = blockIdx.x * size_t(blockDim.x) + threadIdx.x;
    if (threadID >= numABRs) return;

    const ABR &abr = abrs[threadID];
    range1f valueRange = abr.valueRange;

    if (valueRange.upper < valueRange.lower) {
      abrMaxOpacities[threadID] = 0.f;
      return;
    }

    valueRange.lower -= xfRange.lower;
    valueRange.lower /= xfRange.upper-xfRange.lower;
    valueRange.upper -= xfRange.lower;
    valueRange.upper /= xfRange.upper-xfRange.lower;

    int lo = clamp(int(valueRange.lower*(numColors-1)),  0,(int)numColors-1);
    int hi = clamp(int(valueRange.upper*(numColors-1))+1,0,(int)numColors-1);

    float maxOpacity = 0.f;
    for (int i=lo; i<=hi; ++i) {
      maxOpacity = fmaxf(maxOpacity,colorMap[i].w);
    }

    abrMaxOpacities[threadID] = maxOpacity;
  }

  void ExaBrickModel::computeMaxOpacities(OWLContext owl,
                                          OWLBuffer colorMap,
                                          range1f xfRange)
  {
#if EXA_STITCH_EXA_BRICK_TRAVERSAL_MODE == MC_DDA_TRAVERSAL || EXA_STITCH_EXA_BRICK_TRAVERSAL_MODE == MC_BVH_TRAVERSAL
    if (grid && grid->dims != vec3i(0)) {
      grid->computeMaxOpacities(owl,colorMap,xfRange);
    }
#endif

#if  EXA_STITCH_EXA_BRICK_SAMPLER_MODE == EXA_BRICK_SAMPLER_ABR_BVH || \
     EXA_STITCH_EXA_BRICK_TRAVERSAL_MODE == EXABRICK_ABR_TRAVERSAL
    {
      size_t numABRs = owlBufferSizeInBytes(abrBuffer)/sizeof(ABR);
      size_t numColors = owlBufferSizeInBytes(colorMap)/sizeof(vec4f);

      size_t numThreads = 1024;
      computeMaxOpacitiesForABRs<<<iDivUp(numABRs, numThreads), numThreads>>>(
        (float *)owlBufferGetPointer(abrMaxOpacities,0),
        (const ABR *)owlBufferGetPointer(abrBuffer,0),
        (const vec4f *)owlBufferGetPointer(colorMap,0),
        numABRs,numColors,xfRange);
    }

    owlGroupBuildAccel(abrBlas);
    owlGroupBuildAccel(abrTlas);
#endif

#if EXA_STITCH_EXA_BRICK_SAMPLER_MODE == EXA_BRICK_SAMPLER_EXT_BVH || \
    EXA_STITCH_EXA_BRICK_TRAVERSAL_MODE == EXABRICK_BVH_TRAVERSAL || \
    EXA_STITCH_EXA_BRICK_TRAVERSAL_MODE == EXABRICK_KDTREE_TRAVERSAL
    {
      size_t numColors = owlBufferSizeInBytes(colorMap)/sizeof(vec4f);

      size_t numThreads = 1024;
      computeMaxOpacitiesForBricks<<<iDivUp(bricks.size(), numThreads), numThreads>>>(
        (float *)owlBufferGetPointer(brickMaxOpacities,0),
        (const range1f *)owlBufferGetPointer(brickValueRanges,0),
        (const vec4f *)owlBufferGetPointer(colorMap,0),
        bricks.size(),numColors,xfRange);
    }

#if EXA_STITCH_EXA_BRICK_SAMPLER_MODE == EXA_BRICK_SAMPLER_EXT_BVH
    owlGroupBuildAccel(extBlas);
    owlGroupBuildAccel(extTlas);
#endif

#if EXA_STITCH_EXA_BRICK_TRAVERSAL_MODE == EXABRICK_BVH_TRAVERSAL
    owlGroupBuildAccel(brickBlas);
    owlGroupBuildAccel(brickTlas);
#endif

#endif
  }
} // ::exa

// vim: sw=2:expandtab:softtabstop=2:ts=2:cino=\:0g0t0
